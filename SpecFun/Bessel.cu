#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include "CPUfunctions.h"
#include "GPUfunctions.h"
#include <iostream>

/// <summary>
/// ��� ����� ���� GPU
/// </summary>
/// <param name="x"> �������� ��������� </param>
/// <param name="v"> ������� ������� </param>
/// <param name="gamma"> �������� ����� ������� �� (v+1) </param>
/// <param name="result"> ���������� �������� </param>
__global__ void BesselOneThread(const double* x, const double v, const double gamma, double* result, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double eps = 1E-12;
    double aNext;
    double diff;
    int k;
    double aprev;
    double summ;
    while (i < N)
    {
        k = 0;
        aprev = 1 / gamma;
        summ = aprev;
        do {
            aNext = -x[i] * x[i] * aprev / ((k + 1) * (v + k + 1) * 4);
            summ += aNext;
            diff = abs(aprev - aNext);
            aprev = aNext;
            k++;
        } while (diff > eps);
        result[i] = summ * pow(x[i] * 0.5, v);
        i += blockDim.x * gridDim.x;
    }
}

/// <summary>
/// ���������� ������� ������� �� ���������� NVidia
/// </summary>
/// <param name="x"> �������� ��������� </param>
/// <param name="v"> ������� ������� </param>
/// <param name="result"> ���������� �������� </param>
/// <param name="size"> ���������� ����� </param>
hipError_t BesselWithCuda(const double* x, const double v, double* result, const unsigned int size)
{
    double* dev_x = 0;
    double* dev_res = 0;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_res, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    double gamma = Gamma(v + 1);
    BesselOneThread << <(size+127)/128, 128 >> > (dev_x, v, gamma, dev_res, size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result, dev_res, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_res);
    hipFree(dev_x);

    return cudaStatus;
}