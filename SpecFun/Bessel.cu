#include "hip/hip_runtime.h"
#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include "CPUfunctions.h"
#include "GPUfunctions.h"
#include <iostream>

/// <summary>
/// ��� ����� ���� GPU
/// </summary>
/// <param name="x"> �������� ��������� </param>
/// <param name="v"> ������� ������� </param>
/// <param name="gamma"> �������� ����� ������� �� (v+1) </param>
/// <param name="result"> ���������� �������� </param>
__global__ void BesselOneThread(const double v, const double* x, const double gamma, double* result, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double eps = 1E-12;
    double aNext;
    double diff;
    int k;
    double aprev;
    double summ;
    while (i < N)
    {
        k = 0;
        aprev = 1 / gamma;
        summ = aprev;
        do {
            aNext = -x[i] * x[i] * aprev / ((k + 1) * (v + k + 1) * 4);
            summ += aNext;
            diff = abs(aprev - aNext);
            aprev = aNext;
            k++;
        } while (diff > eps);
        result[i] = summ * pow(x[i] * 0.5, v);
        i += blockDim.x * gridDim.x;
    }
}

/// <summary>
/// ��� ������� ���������� ������� �������
/// </summary>
/// <param name="x"> �������� ��������� </param>
/// <param name="v"> ������� ������� </param>
/// <param name="gamma"> �������� ����� ������� �� (v+1) </param>
/// <param name="result"> ���������� �������� </param>
__global__ void Jnew(const double v, const double* x, const double gamma, double* result, int N)
{
    __shared__ double p[256];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double eps = 1E-12;
    double aNext;
    double diff;
    double aPrev;
    double a_0;
    if (i < 256)
        p[threadIdx.x] = -1 / (4 * (v + threadIdx.x + 1) * (threadIdx.x + 1));
    __syncthreads();
    while (i < N)
    {
        a_0 = 1 / gamma;
        aPrev = a_0;
        result[i] = a_0;
        int k = 0;
        do {
            aNext = p[k] * aPrev * x[i] * x[i];
            result[i] += aNext;
            diff = abs(aPrev - aNext);
            aPrev = aNext;
            k++;
        } while (diff > eps);
        result[i] *= pow(x[i] * 0.5, v);
        i += blockDim.x * gridDim.x;
    }
}
hipError_t BesselWithCudaNew(const double v, const double* x, double* result, const unsigned int size)
{
    double* dev_x = 0;
    double* dev_res = 0;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_res, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    double gamma = Gamma(v + 1);
    Jnew << <(size + 255) / 256, 256 >> > (v, dev_x, gamma, dev_res, size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result, dev_res, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_res);
    hipFree(dev_x);

    return cudaStatus;
}

hipError_t BesselWithCuda(const double v, const double* x, double* result, const unsigned int size)
{
    double* dev_x = 0;
    double* dev_res = 0;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_res, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    double gamma = Gamma(v + 1);
    BesselOneThread << <(size+127)/128, 128 >> > (v, dev_x, gamma, dev_res, size);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result, dev_res, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_res);
    hipFree(dev_x);

    return cudaStatus;
}