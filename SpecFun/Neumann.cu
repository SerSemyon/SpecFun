#include "hip/hip_runtime.h"
#pragma once
#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include "CPUfunctions.h"
#include "GPUfunctions.h"
#include <iostream>

/// <summary>
/// ��� ����� ���� GPU
/// </summary>
/// <param name="x"> �������� ��������� </param>
/// <param name="v"> ������� ������� </param>
/// <param name="gamma"> �������� ����� ������� �� (v+1) </param>
/// <param name="result"> ���������� �������� </param>
__global__ void Y0_OneThread(const double* const x, double* result, int size, const double* const J0)
{
    const double C = 0.5772156;
    const double b0[] = {
        -0.02150'51114'49657'55061,
        -0.27511'81330'43518'79146,
        0.19860'56347'02554'15556,
        0.23425'27461'09021'80210,
        -0.16563'59817'13650'41312,
        0.04462'13795'40669'28217,
        -0.00693'22862'91523'18829,
        0.00071'91174'03752'30309,
        -0.00005'39250'79722'93939,
        0.00000'30764'93288'10848,
        -0.00000'01384'57181'23009,
        0.00000'00050'51054'36909,
        -0.00000'00001'52582'85043,
        0.00000'00000'03882'86747,
        -0.00000'00000'00084'42875,
        0.00000'00000'00001'58748,
        -0.00000'00000'00000'02608,
        0.00000'00000'00000'00038
    };
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < size)
    {
        double T2 = x[i] / 8.0;
        T2 = 2.0 * T2 * T2 - 1.0;
        double T_previous = 1.0; double T_current = T2;
        double T;
        double sum = b0[0] * T_previous + b0[1] * T_current;
        for (int n = 2; n <= 17; n++) {
            T = 2.0 * T2 * T_current - T_previous;
            sum += b0[n] * T;
            T_previous = T_current; T_current = T;
        };
        sum += (log(x[i] / 2.0) + C) * J0[i] * 2.0 / M_PI;
        result[i] = sum;
        i += blockDim.x * gridDim.x;
    }
}

void Y0_CUDA(const double* const x, double* result, const unsigned int size, const double* const J0)
{
    double* dev_x = 0;
    double* dev_res = 0;

    /*hipMalloc((void**)&dev_res, size * sizeof(double));
    hipMalloc((void**)&dev_x, size * sizeof(double));
    hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);*/



    hipError_t cudaStatus;
    
    cudaStatus = hipMalloc((void**)&dev_res, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    
    cudaStatus = hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Y0_OneThread << <(size + 127) / 128, 128 >> > (dev_x, dev_res, size, J0);

    /*hipError_t cudaStatus = hipGetLastError();
    hipDeviceSynchronize();
    hipMemcpy(result, dev_res, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_res);
    hipFree(dev_x);*/

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    
    cudaStatus = hipMemcpy(result, dev_res, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    
    Error:
        hipFree(dev_res);
        hipFree(dev_x);
}