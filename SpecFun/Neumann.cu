#include "hip/hip_runtime.h"
#pragma once
#define _USE_MATH_DEFINES
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include "CPUfunctions.h"
#include "GPUfunctions.h"
#include <iostream>

/// <summary>
/// ��� ����� ���� GPU
/// </summary>
/// <param name="x"> �������� ��������� </param>
/// <param name="v"> ������� ������� </param>
/// <param name="gamma"> �������� ����� ������� �� (v+1) </param>
/// <param name="result"> ���������� �������� </param>
__global__ void Y0_OneThread(const double* const x, double* result, int size, const double* const J0)
{
    const double C = 0.5772156;
    const double b0[] = {
        -0.02150'51114'49657'55061,
        -0.27511'81330'43518'79146,
         0.19860'56347'02554'15556,
         0.23425'27461'09021'80210,
        -0.16563'59817'13650'41312,
         0.04462'13795'40669'28217,
        -0.00693'22862'91523'18829,
         0.00071'91174'03752'30309,
        -0.00005'39250'79722'93939,
         0.00000'30764'93288'10848,
        -0.00000'01384'57181'23009,
         0.00000'00050'51054'36909,
        -0.00000'00001'52582'85043,
         0.00000'00000'03882'86747,
        -0.00000'00000'00084'42875,
         0.00000'00000'00001'58748,
        -0.00000'00000'00000'02608,
         0.00000'00000'00000'00038
    };
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < size)
    {
        double T2 = x[i] / 8.0;
        T2 = 2.0 * T2 * T2 - 1.0;
        double T_previous = 1.0; double T_current = T2;
        double T;
        double sum = b0[0] * T_previous + b0[1] * T_current;
        for (int n = 2; n <= 17; n++) {
            T = 2.0 * T2 * T_current - T_previous;
            sum += b0[n] * T;
            T_previous = T_current; T_current = T;
        };
        sum += (log(x[i] / 2.0) + C) * J0[i] * 2.0 / M_PI;
        result[i] = sum;
        i += blockDim.x * gridDim.x;
    }
}

void Y0_CUDA(const double* const x, double* result, const unsigned int size, const double* const J0)
{
    double* dev_x = 0;
    double* dev_res = 0;
    double* dev_J0 = 0;

    hipMalloc((void**)&dev_x, size * sizeof(double));
    hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_res, size * sizeof(double));

    hipMalloc((void**)&dev_J0, size * sizeof(double));
    hipMemcpy(dev_J0, J0, size * sizeof(double), hipMemcpyHostToDevice);

    Y0_OneThread << <(size + 127) / 128, 128 >> > (dev_x, dev_res, size, dev_J0);

    hipGetLastError();
    hipDeviceSynchronize();

    hipMemcpy(result, dev_res, size * sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(dev_res);
    hipFree(dev_x);
    hipFree(dev_J0);
}


/// <summary>
/// ��� ����� ���� GPU
/// </summary>
/// <param name="x"> �������� ��������� </param>
/// <param name="v"> ������� ������� </param>
/// <param name="gamma"> �������� ����� ������� �� (v+1) </param>
/// <param name="result"> ���������� �������� </param>
__global__ void Y1_OneThread(const double* const x, double* result, int size, const double* const J1)
{
    const double C = 0.5772156;
    const double b1[] = {
    -0.04017'29465'44414'07579,
    -0.44444'71476'30558'06261,
    -0.02271'92444'28417'73587,
    0.20664'45410'17490'51976,
    -0.08667'16970'56948'52366,
    0.01763'67030'03163'13441,
    -0.00223'56192'94485'09524,
    0.00019'70623'02701'54078,
    -0.00001'28858'53299'24086,
    0.00000'06528'47952'35852,
    -0.00000'00264'50737'17479,
    0.00000'00008'78030'11712,
    -0.00000'00000'24343'27870,
    0.00000'00000'00572'61216,
    -0.00000'00000'00011'57794,
    0.00000'00000'00000'20347,
    -0.00000'00000'00000'00314,
    0.00000'00000'00000'00004
    };
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    while (i < size)
    {
        double z = x[i] / 8.0;
        double T_previous = 1.0, T_current = z;
        double T;
        double s = b1[0] * T_current;
        for (int n = 1; n <= 17; n++) {
            T = 2.0 * z * T_current - T_previous;
            T_previous = T_current; T_current = T;
            T = 2.0 * z * T_current - T_previous;
            s += b1[n] * T;
            T_previous = T_current; T_current = T;
        };
        s += (C + log(x[i] / 2.0)) * J1[i] * 2.0 / M_PI - 2.0 / (M_PI * x[i]);
        result[i] = s;
        i += blockDim.x * gridDim.x;
    }
}

void Y1_CUDA(const double* const x, double* result, const unsigned int size, const double* const J1)
{
    double* dev_x = 0;
    double* dev_res = 0;
    double* dev_J1 = 0;

    hipMalloc((void**)&dev_x, size * sizeof(double));
    hipMemcpy(dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_res, size * sizeof(double));

    hipMalloc((void**)&dev_J1, size * sizeof(double));
    hipMemcpy(dev_J1, J1, size * sizeof(double), hipMemcpyHostToDevice);

    Y1_OneThread << <(size + 127) / 128, 128 >> > (dev_x, dev_res, size, dev_J1);

    hipGetLastError();
    hipDeviceSynchronize();

    hipMemcpy(result, dev_res, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_res);
    hipFree(dev_x);
    hipFree(dev_J1);
}