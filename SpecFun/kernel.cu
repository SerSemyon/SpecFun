﻿#pragma once
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include "CPUfunctions.h"
#include "GPUfunctions.h"
#include "Test.h"
#include "log_duration.h"

int main()
{
    TestJ0();
    TestBesselCuda();
    TestChebyshevPolynomials();
}